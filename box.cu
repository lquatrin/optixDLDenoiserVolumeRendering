#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// Box attributes
rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );

// Current ray state
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Atribute to be used in closest_hit_radiance0
/**
 * . Attributes MUST be modified between rtPotentialIntersection and rtReportIntersection calls.
**/
rtDeclareVariable(float3, shadingnormal, attribute shadingnormal, );
rtDeclareVariable(float2, tvalue, attribute tvalue, );

static __device__ float3 boxnormal (float t, float3 t0, float3 t1)
{
  float3 neg = make_float3(t == t0.x ? 1 : 0,
                           t == t0.y ? 1 : 0,
                           t == t0.z ? 1 : 0);

  float3 pos = make_float3(t == t1.x ? 1 : 0,
                           t == t1.y ? 1 : 0,
                           t == t1.z ? 1 : 0);
  
  return pos - neg;
}

/**
 * . Attribute Variables may only be set in between these two function calls. This
 * ensures that the Variables always represent the values of the closest hit yet found.
 * If the Variables are read inside the Closest Hit Program it is guaranteed, that they
 * represent the values of the closest intersection.
 *
 * . Typically Attribute Variables are used to communicate intersection specific information 
 * to the Closest Hit or Any Hit Programs such as surface normal vectors or texture coordinates.
**/
RT_PROGRAM void box_intersect (int primIdx)
{
  float3 t0   = (boxmin - ray.origin) / ray.direction;
  float3 t1   = (boxmax - ray.origin) / ray.direction;
  float3 near = fminf(t0, t1);
  float3 far  = fmaxf(t0, t1);
  float tmin  = fmaxf(near);
  float tmax  = fminf(far);


  if(tmin <= tmax)
  {
    bool check_second = true;
    // returns true if the t-value lies inside the allowed range of the ray
    if (rtPotentialIntersection(tmin))
    {
       tvalue = make_float2(tmin, tmax);
       shadingnormal = boxnormal(tmin, t0, t1);
       // is called with the material index 
       // what material should be used on this part of the geometry
       if (rtReportIntersection(0))
         check_second = false;
    } 
    if(check_second)
    {
      if (rtPotentialIntersection(tmax))
      {
        tvalue = make_float2(tmin, tmax);
        shadingnormal = boxnormal(tmax, t0, t1);
        rtReportIntersection(0);
      }
    }
  }
}

/**
 * . Geometry objects also need a Bounding Box Program assigned. It must return an
 * axis aligned bounding box that fully encloses the primitive at the primitive index
 * given as an argument. It is used by the Acceleration Structures while traversing
 * and for building the acceleration tree. While a fast implementation is desirable, the
 * accuracy of the bounding box is also important to build good quality acceleration
 * trees. Accurate means that the bounding box should be as small as possible while
 * still fully enclosing the primitive.
**/
RT_PROGRAM void box_bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}
