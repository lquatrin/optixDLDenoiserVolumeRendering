#include "hip/hip_runtime.h"
#include "tutorial.h"
#include "random.h"

// Internally providade variables from OptiX
rtDeclareVariable(uint2,               launch_index, rtLaunchIndex, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload    , );
rtDeclareVariable(optix::Ray,          ray,          rtCurrentRay , );

// Atribute set on box and parallelogram
rtDeclareVariable(float3, shadingnormal, attribute shadingnormal, );
rtDeclareVariable(float2,        tvalue, attribute tvalue       , );

rtDeclareVariable(float,    scene_epsilon, , );
rtDeclareVariable(rtObject, top_object   , , );

// Pinhole camera implementation
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(Matrix3x3, normal_matrix, , );
rtDeclareVariable(float3, VolSizes, , );

rtDeclareVariable(unsigned int, frame_number, , );

rtBuffer<float4, 2> out_render_buffer;
rtBuffer<float4, 2> out_albedo_buffer;
rtBuffer<float4, 2> out_normal_buffer;

rtTextureSampler<float, 3> TexVolume;
rtTextureSampler<float4, 1> TexTransferFunction;
rtTextureSampler<float4, 3> TexGradientVolume;

RT_PROGRAM void RayGenerationProgram ()
{
  size_t2 screen = out_render_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon );

  unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;
  prd.alpha_ray = 0.0f;
  prd.normal_output = make_float3(0.0f, 0.0f, 0.0f);
  prd.normal_total_weight = 0.0f;
  prd.albedo_output = make_float3(0.0f, 0.0f, 0.0f);
  prd.id_noise = frame_number % 1024;
  prd.seed = seed;

  // trace a ray giving the object hierarchy group that will be traced
  rtTrace(top_object, ray, prd);

  // Only update the current state of the output buffer
  if (frame_number > 1)
  {
    //*
    float a = 1.0f / (float)frame_number;

    // update output normalized color
    float3 normalized_color = prd.result * prd.alpha_ray + make_float3(0.0f, 0.0f, 0.0f) * (1.0f - prd.alpha_ray);
    float4 curr_color = make_float4(normalized_color.x, normalized_color.y, normalized_color.z, prd.alpha_ray);
    float4 old_color_f4 = out_render_buffer[launch_index];
    out_render_buffer[launch_index] = lerp(old_color_f4, curr_color, a);
    
    // update output normal
    float4 curr_normal = make_float4(prd.normal_output / prd.normal_total_weight, prd.alpha_ray);
    float4 old_normal_f4 = out_normal_buffer[launch_index];
    out_normal_buffer[launch_index] = lerp(old_normal_f4, curr_normal, a);

    // update output albedo
    float4 curr_albedo = make_float4(prd.albedo_output.x, prd.albedo_output.y, prd.albedo_output.z, prd.alpha_ray);
    float4 old_albedo_f4 = out_albedo_buffer[launch_index];
    out_albedo_buffer[launch_index] = lerp(old_albedo_f4, curr_albedo, a);
    // */
  }
  // First frame!
  else
  {
    // output normalized color
    float3 normalized_color = prd.result * prd.alpha_ray + make_float3(0.0f, 0.0f, 0.0f) * (1.0f - prd.alpha_ray);
    out_render_buffer[launch_index] = make_float4(normalized_color.x, normalized_color.y, normalized_color.z, prd.alpha_ray);

    // The current normal is the amount of normals averaged by their weights
    float3 normals_res = prd.normal_output / prd.normal_total_weight;
    out_normal_buffer[launch_index] = make_float4(normals_res.x, normals_res.y, normals_res.z, prd.alpha_ray);

    // Color without the gradient shading
    out_albedo_buffer[launch_index] = make_float4(prd.albedo_output.x, prd.albedo_output.y, prd.albedo_output.z, prd.alpha_ray);
  }
}

/**
 * If the ray misses and finds no intersection. we may implement the background.
**/
// Returns solid color for miss rays
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss ()
{
  prd_radiance.result = bg_color;
}
 
const int NUMBER_OF_CASTED_RAYS = 1;
const float DO_INITIAL_STEP = 2.0f;
const float DO_MAX_DISTANCE = 80.0f;
float EvaluateDirectionalOcclusion (float3 Wpos)
{
  float3 VolBBMin = -VolSizes * 0.5f;
  float3 VolBBMax =  VolSizes * 0.5f;

  float S_Vt = 0.0;
  float S_Wt = 0.0;

  int iray = 0;
  while (iray < NUMBER_OF_CASTED_RAYS)
  {
    // get noise values for direction [0...1, 
    float3 sn = make_float3(rnd(prd_radiance.seed), (rnd(prd_radiance.seed) * 2.0f) - 1.0f, (rnd(prd_radiance.seed) * 2.0f) - 1.0f);

    float3 fake_light_direction = normalize(-W * sn.z + U * sn.y + V * sn.x);
    
    // Let's evaluate shadow
    float Vt = 1.0f;
    float s = DO_INITIAL_STEP;
    while (s < DO_MAX_DISTANCE)
    {
      float d = min(0.5f, DO_MAX_DISTANCE - s);
    
      // Check if we're out of the bounding box
      float3 spos = Wpos + fake_light_direction * s;

      if (!(spos.x > VolBBMin.x && spos.x < VolBBMax.x &&
            spos.y > VolBBMin.y && spos.y < VolBBMax.y &&
            spos.z > VolBBMin.z && spos.z < VolBBMax.z))
        break;
          
      float3 txpos = (spos + (VolSizes * 0.5f)) / VolSizes;
      float st = tex1D(TexTransferFunction, tex3D(TexVolume, txpos.x, txpos.y, txpos.z)).w;

      Vt = Vt * exp(-st * d);
      if (Vt < 0.05) break;

      s = s + d;
    }

    float wtc = dot(normalize(-W), fake_light_direction);

    S_Vt = S_Vt + Vt * wtc;
    S_Wt = S_Wt + wtc;
    
    iray = iray + 1;
    prd_radiance.id_noise = prd_radiance.id_noise + 1;
  }
  
  return S_Vt / S_Wt;
}

const bool apply_phong = true;
float3 ShadeWithPhong (float3 Li, float ka, float kd, float ks, float3 pos, float expt, float3 Wpos, float ta)
{
  float3 WorldLightingPos = eye;
  float4 tex_gt_n = tex3D(TexGradientVolume, pos.x, pos.y, pos.z);
  
  if (apply_phong)
  {
    float3 gradient_normal = make_float3(tex_gt_n.x, tex_gt_n.y, tex_gt_n.z);
    
    if (gradient_normal.x != 0 && gradient_normal.y != 0 && gradient_normal.z != 0)
    {
      gradient_normal = normalize(gradient_normal);
    
      float3 light_direction = normalize(WorldLightingPos - Wpos);
      float3 eye_direction = normalize(eye - Wpos);
      float3 halfway_vector = normalize(eye_direction + light_direction);
      
      float dot_diff = max(0.0f, dot(gradient_normal, light_direction));
      float dot_spec = max(0.0f, dot(halfway_vector, gradient_normal));

      float3 Is = make_float3(1, 1, 1);
      float Nshininess = 20.0f;
      
      float clr_norm = (1.0 / (ka + kd + ks));
      Li = clr_norm * (Li * ka * ta + Li * kd * dot_diff  + ks * Is * pow(dot_spec, Nshininess));
    }
  }
  else
  {
    Li = Li * ta;
  }

  /////////////////////////////////////////////////////////////////////////////
  // Accumulate gradient
  float3 normal_g = make_float3(tex_gt_n.x, tex_gt_n.y, tex_gt_n.z);
  float3 normal_eyespace = (length(normal_g) > 0.f) ? normalize(normal_matrix * normal_g) : make_float3(0., 0., 1.);
  prd_radiance.normal_output += normal_eyespace * (1.0 - expt);
  prd_radiance.normal_total_weight += (1.0 - expt);

  return Li;
}

// Returns shading normal as the surface shading result
/**
 * . The intersection details such as texture coordinates should be 
 * communicated to the Closest Hit Program via Attribute Variables 
 * that were calculated in the intersection program.
 *
 * . rtTrace may be recursively called to implement shadows or reflection.
 *
 * . result may be stored in the payload of the ray.
 *
 * . Apply ray marching here!
**/
const float Ka = 0.60f;
const float Kd = 0.15f;
const float Ks = 0.25f;
RT_PROGRAM void closest_hit_radiance0 ()
{
  float3 r0 = ray.origin;
  float3 rd = ray.direction;

  float step_pos = rnd(prd_radiance.seed);// ((frame_number * 10) % 100) * 0.01;
  float tmin_pos_epsilon = 0.0001f;
  float3 pt = (r0 + rd * (tvalue.x + tmin_pos_epsilon));

  float integration_step = 0.5f;

  float3 L = make_float3(0.0f);
  float3 Lalbedo = make_float3(0.0f);
  float T = 1.0f;

  float s = tvalue.x;
  while (s < tvalue.y)
  {
    float d = min(integration_step, tvalue.y - s);

    float3 Wpos = r0 + rd * (s + d * step_pos);
    float3 pos = (Wpos + (VolSizes * 0.5f)) / VolSizes;

    float4 Ltex = tex1D(TexTransferFunction, tex3D(TexVolume, pos.x, pos.y, pos.z));
    float3 Li = make_float3(Ltex.x, Ltex.y, Ltex.z);
    float expt = exp(-Ltex.w * d);
    
    if (Ltex.w > 0.0f)
    {
      float S_Vt = EvaluateDirectionalOcclusion(Wpos);
      Li = ShadeWithPhong(Li, Ka, Kd, Ks, pos, expt, Wpos, S_Vt);
    }

    /////////////////////////////////////////////////////////////////////////////
    // Accumulate albedo
    Lalbedo = Lalbedo + T * (1.0 - expt) * make_float3(Ltex.x, Ltex.y, Ltex.z);

    L = L + T * (1.0 - expt) * Li;
    T = T * expt;
    if ((1.0 - T) > 0.95) break;

    s = s + d;
  }

  prd_radiance.result = L;
  prd_radiance.alpha_ray = 1.0 - T;

  prd_radiance.albedo_output = Lalbedo;
}

// any hit program
/**
 * . as opposed to the Closest Hit Program, it might be
 * called multiple times for a single ray cast.
 *
 * . The intersections for which the program is executed may not be ordered 
 * along the ray, but eventually all intersections can be enumerated by 
 * calling rtIgnoreIntersection on each of them.
 *
 * . The Any Hit Program can be used if the application requires to perform 
 * actions at each surface intersection. Rays can also be terminated with 
 * rtTerminateRay, then * the trace ends and no further executions of the Any 
 * Hit Program may be invoked. This can be used if only the knowledge whether
 * the ray hits anything or not is needed, for example in shadow rays.
**/
// Set pixel to solid color upon failur
RT_PROGRAM void exception ()
{
  out_render_buffer[launch_index] = make_float4(bad_color.x, bad_color.y, bad_color.z, 1.0f);
}
